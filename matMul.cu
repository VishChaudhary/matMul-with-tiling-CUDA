#include "hip/hip_runtime.h"
#define TILE_WIDTH 16

__global__ void matrixMulKernel (float* M, float * N, float* P, int width){
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x;  int ty = treadIdx.y;

  // Identify the row and column of the P element to work on
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  //Loop over the M and N tiles required to compute P element
  float Pvalue = 0;

  for( int ph = 0; ph<(width/(float)TILE_WIDTH); ++ph){
    if ((row < width) && (ph*TILE_WIDTH) < width){
      Mds[ty][tx] = M[row*width + ph*TILE_WIDTH +tx];
    }
    else {
      Mds[ty][tx] = 0.0f;
    }

    if ((ph*TILE_WIDTH+ty) < width && col < width){
      Nds[ty][tx] = N[(ph*TILE_WIDTH +ty)*width + col];
    }
    else Nds[ty][tx] = 0.0f;
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  if((row<width) && (col<width)){
    P[row*width+col] = Pvalue;
  }
}
